#include "hip/hip_runtime.h"
#include <stdio.h>

//cuda headers
#include <hip/hip_runtime.h>
#include <helper-timer.h>

//macros
#define BLOCK_WIDTH 32

//global variables
int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// cuda kernel function
__global__ void matMulGPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    //variable declarations
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    // code
    if((row < numARows) && (column < numBColumns))
    {
        int value = 0.0;
        
        for(int k = 0; k < numAColumns; k++)
        {
            int a = A[row * numAColumns + k];
            int b = B[k * numBColumns + column];
            value += a * b;
        }

        C[row * numCColumns + column] = value;
    }

}

int main()
{
    // function declarations
    void InitA(int *data,int,int);
    void InitB(int *data,int,int);
    void matmulCPU(int *, int *, int *, int, int, int, int);
    void cleanup(void);

    // variable declaration
    int numARows = BLOCK_WIDTH;
    int numAColumns = BLOCK_WIDTH;
    int numBRows = BLOCK_WIDTH;
    int numBColumns = BLOCK_WIDTH;
    int numCRows = numARows;
    int numCColumns = numBColumns;

    int numGoldRows = numARows;
    int numGodlColumns = numBColumns;

    int sizeA = numARows * numAColumns * sizeof(int);
    int sizeB = numBRows * numBColumns * sizeof(int);
    int sizeC = numCRows * numBColumns * sizeof(int);
    int sizeGold = numGoldRows * numGodlColumns * sizeof(int);

    hipError_t result = hipSuccess;

    //code
    //host memory allocation
    hostA = (int*)malloc(sizeA);
    if (hostA == NULL)
    {
        printf("Host Memory allocation is failed for hostA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostB = (int*)malloc(sizeB);
    if (hostB == NULL)
    {
        printf("Host Memory allocation is failed for hostB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostC = (int*)malloc(sizeC);
    if (hostC == NULL)
    {
        printf("Host Memory allocation is failed for hostC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    gold = (int*)malloc(sizeGold);
    if (gold == NULL)
    {
        printf("Host Memory allocation is failed for gold matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // printing matrix dimensions and sizes
    printf("The Dimensions of Matrix 'hostA' are : %d x %d\n",numARows,numAColumns);
    printf("The Dimensions of Matrix 'hostB' are : %d x %d\n",numBRows,numBColumns);
    printf("The Dimensions of Matrix 'hostC' are : %d x %d\n",numCRows,numCColumns);
    printf("The Dimensions of Matrix 'gold' are : %d x %d\n",numGoldRows,numGodlColumns);

    printf("Size of Matrix hostA = %d\n",sizeA);
    printf("Size of Matrix hostB = %d\n",sizeB);
    printf("Size of Matrix hostC = %d\n",sizeC);
    printf("Size of Matrix gold = %d\n",gold);
}